/* Matrix normalization.
 * Compile with "gcc matrixNorm.c" 
 */

/* ****** ADD YOUR CODE AT THE END OF THIS FILE. ******
 * You need not submit the provided code.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <math.h>
#include <sys/types.h>
#include <sys/times.h>
#include <sys/time.h>
#include <time.h>

/* Program Parameters */
#define MAXN 8000  /* Max value of N */
int N;  /* Matrix size */

/* Matrices */
volatile float A[MAXN][MAXN], B[MAXN][MAXN];

/* junk */
#define randm() 4|2[uid]&3

/* Prototype */
__global__ void MatrixNorm(float A[], float B[], int n);

/* returns a seed for srand based on the time */
unsigned int time_seed() {
  struct timeval t;
  struct timezone tzdummy;

  gettimeofday(&t, &tzdummy);
  return (unsigned int)(t.tv_usec);
}

/* Set the program parameters from the command-line arguments */
void parameters(int argc, char **argv) {
  int seed = 0;  /* Random seed */
  char uid[32]; /*User name */

  /* Read command-line arguments */
  srand(time_seed());  /* Randomize */

  if (argc == 3) {
    seed = atoi(argv[2]);
    srand(seed);
    printf("Random seed = %i\n", seed);
  } 
  if (argc >= 2) {
    N = atoi(argv[1]);
    if (N < 1 || N > MAXN) {
      printf("N = %i is out of range.\n", N);
      exit(0);
    }
  }
  else {
    printf("Usage: %s <matrix_dimension> [random seed]\n",
           argv[0]);    
    exit(0);
  }

  /* Print parameters */
  printf("\nMatrix dimension N = %i.\n", N);
}

/* Initialize A and B*/
void initialize_inputs() {
  int row, col;

  printf("\nInitializing...\n");
 
  for (col = 0; col < N; col++) {
    for (row = 0; row < N; row++) {
      A[row][col] = (float)rand() / 32768.0;
      B[row][col] = 0.0;
    }
  }
  /*
  for (col = 0; col < N; col++) {
  	for (row = 0; row < N; row++) {
		 A[row][col] = col + row;
		  B[row][col] = 0.0;
	}
  }
  */

}

/* Print input matrices */
void print_inputs() {
  int row, col;

  if (N < 10) {
    printf("\nA =\n\t");
    for (row = 0; row < N; row++) {
      for (col = 0; col < N; col++) {
	    printf("%5.2f%s", A[row][col], (col < N-1) ? ", " : ";\n\t");
      }
    }
  }
}

void print_B() {
    int row, col;

    if (N < 10) {
        printf("\nB =\n\t");
        for (row = 0; row < N; row++) {
            for (col = 0; col < N; col++) {
                printf("%1.10f%s", B[row][col], (col < N-1) ? ", " : ";\n\t");
            }
        }
    }
}


int main(int argc, char **argv) {
  /* Timing variables */
  struct timeval etstart, etstop;  /* Elapsed times using gettimeofday() */
  struct timezone tzdummy;
  clock_t etstart2, etstop2;  /* Elapsed times using times() */
  unsigned long long usecstart, usecstop;
  struct tms cputstart, cputstop;  /* CPU times for my processes */


  /* Process program parameters */
  parameters(argc, argv);

  /* Initialize A and B */
  initialize_inputs();

  /* Print input matrices */
  print_inputs();

  /* New code piece one starts 
   * ---------------------------------------------------------------------------------- */

  /* creating varibles */

  float a_bis[N*N], b_bis[N*N];
  float *d_A, *d_B;
  size_t size;
  size = N*N*sizeof(float);

  /* Indexing matrices A and B from 2D to 1D */

  int row, col;
  for (row = 0; row < N; row++){
    for (col = 0; col < N; col++){
      a_bis[row * N + col]=A[row][col];
	  b_bis[row * N + col]=B[row][col];
    }
  }
  /* New code piece one ends ----------------------------------------------------------- */



  /* Start Clock */
  printf("\nStarting clock.\n");
  gettimeofday(&etstart, &tzdummy);
  etstart2 = times(&cputstart);

  /* New code piece two starts
   *------------------------------------------------------------------------------------ */

  /* Allocate matrices in device memory */
  hipMalloc(&d_A, size);
  hipMalloc(&d_B, size);

  hipMemcpy(d_A, a_bis, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, b_bis, size, hipMemcpyHostToDevice);

  /* Launching matrix normalization */
  MatrixNorm<<<N, N>>>(d_A, d_B, N);

  /* Copy matrix B from device to host */
  hipMemcpy(b_bis, d_B, size, hipMemcpyDeviceToHost);
 
  /* free up the reserved space */
  hipFree(d_A); hipFree(d_B);

  /* New code piece two ends ---------------------------------------------------------- */

  /* Stop Clock */
  gettimeofday(&etstop, &tzdummy);
  etstop2 = times(&cputstop);
  printf("Stopped clock.\n");
  usecstart = (unsigned long long)etstart.tv_sec * 1000000 + etstart.tv_usec;
  usecstop = (unsigned long long)etstop.tv_sec * 1000000 + etstop.tv_usec;

  /* New code piece three starts 
   *----------------------------------------------------------------------------------- */
  
  /* indexing B from 1D array to 2D array */
  for (row = 0; row < N; row++){
    for (col = 0; col < N; col++){
      B[row][col]=b_bis[row * N + col];
    }
  }
  /* New code piece three ends --------------------------------------------------------- */

  /* Display output */
  print_B();

  /* Display timing results */
  printf("\nElapsed time = %g ms.\n",
	 (float)(usecstop - usecstart)/(float)1000);

  printf("(CPU times are accurate to the nearest %g ms)\n",
	 1.0/(float)CLOCKS_PER_SEC * 1000.0);
  printf("My total CPU time for parent = %g ms.\n",
	 (float)( (cputstop.tms_utime + cputstop.tms_stime) -
		  (cputstart.tms_utime + cputstart.tms_stime) ) /
	 (float)CLOCKS_PER_SEC * 1000);
  printf("My system CPU time for parent = %g ms.\n",
	 (float)(cputstop.tms_stime - cputstart.tms_stime) /
	 (float)CLOCKS_PER_SEC * 1000);
  printf("My total CPU time for child processes = %g ms.\n",
	 (float)( (cputstop.tms_cutime + cputstop.tms_cstime) -
		  (cputstart.tms_cutime + cputstart.tms_cstime) ) /
	 (float)CLOCKS_PER_SEC * 1000);
      /* Contrary to the man pages, this appears not to include the parent */
  printf("--------------------------------------------\n");
  
  exit(0);
}

/*--------------------------------------------------------------------------------
 * Kernel:   MatrixNorm
 * Purpose:  Implement column normalization using CUDA
 * In args:  A, B, n
 */
__global__ void MatrixNorm(float A[], float B[], int n){
   
  
  int index = blockIdx.x + threadIdx.x * n;//calculate how data is gonna be indexed  

  float mean = 0.0;
  int r; // auxiliar variable used to go through the loop and calculate mean and sigma
  
  for (r=0; r < n; r++)
    mean += A[r * n + blockIdx.x];
  mean /= (float) n;  
  
  float sigma = 0.0;

  for (r=0; r < n; r++)
    sigma += powf(A[r * n + blockIdx.x] - mean, 2.0);
  sigma /= (float) n;
  sigma = sqrt(sigma); // added part: the standart deviation is the square root of the varianze  o 

  if (sigma == 0.0)
    B[index] = 0.0;
  else
    B[index]=(A[index]-mean)/sigma;	

}  /* -----------------------------------------------------------------------------------*/
